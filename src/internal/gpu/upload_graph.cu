#include "hip/hip_runtime.h"
#include "cubu/internal/gpu.hpp"

namespace cubu::internal {
std::tuple<linear_resource<glm::vec2>,
           linear_resource<int>,
           linear_resource<float>>
gpu::upload_graph(const graph_t& graph, const glm::uvec2& resolution)
{
  // *** Calculate the edge and point counts
  size_t pointCount = graph.point_count(), edgeCount = graph.edges().size();

  // *** Create a new vector for all the points
  std::vector<glm::vec2> h_points;

  // *** Allocate the memory for all the points and end of line markers
  h_points.reserve(pointCount);

  // *** Create a new vector for all the edge indices
  std::vector<int> h_edgeIndices;

  // *** Allocate the memory for the edge indices
  h_edgeIndices.reserve(edgeCount);

  // *** Create a new vector for all the edge lengths
  std::vector<float> h_edgeLengths;

  // *** Allocate the memory for the edge indices
  h_edgeLengths.reserve(edgeCount);

  // *** Get the range of the graph
  glm::vec2 range = graph.bounds().max - graph.bounds().min;

  // *** Calculate the scale
  float scale = range.x > range.y ? static_cast<float>(resolution.x) / range.x
                                  : static_cast<float>(resolution.y) / range.y;

  // *** Calculate the translation
  glm::vec2 translation = {
    (static_cast<float>(resolution.x) - scale * range.x) / 2,
    (static_cast<float>(resolution.y) - scale * range.y) / 2
  };

  // *** Loop over all the poly lines
  for (const auto& line : graph.edges()) {
    // *** Add the starting point of the next polyline to the list of edge
    // indices
    h_edgeIndices.emplace_back(h_points.size());

    // *** Add the length of the edge to the list of edge lengths
    h_edgeLengths.emplace_back(line->length());

    // *** Loop over all the points in the line
    for (const auto& point : line->points()) {
      h_points.emplace_back((point - graph.bounds().min) * scale + translation);
    }
  }

  // *** Create a linear texture containing all the points and data
  linear_resource pointsRes(
    h_points, hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat));

  // *** Create a linear texture containing the indices of the starting point of
  // each edge in the points texture
  linear_resource edgeIndicesRes(
    h_edgeIndices,
    hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned));

  // *** Create a linear texture containing the lengths of each edge
  linear_resource edgeLengthsRes(
    h_edgeLengths,
    hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat));

  return { std::move(pointsRes),
           std::move(edgeIndicesRes),
           std::move(edgeLengthsRes) };
}
} // namespace cubu::internal