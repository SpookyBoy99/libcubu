#include "hip/hip_runtime.h"
#include "cubu/internal/gpu.hpp"
#include "cubu/internal/gpu_check.hpp"

namespace cubu::internal {
typedef enum convolution_direction
  : char
{
  convolveColumns = 0,
  convolveRows
} convolution_direction_t;

namespace kernels {
__global__ void
generateDensityMapFast(float* densityOutput,
                       hipTextureObject_t pointsTex,
                       hipTextureObject_t edgeIndicesTex,
                       hipTextureObject_t edgeLengthsTex,
                       size_t pointCount,
                       size_t edgeCount,
                       size_t pitch)
{
  // *** Do nothing if no output is specified
  if (!densityOutput) {
    return;
  }

  // *** Get the index and stride from gpu
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = blockDim.x * gridDim.x;

  // *** Loop over the edges
  for (size_t i = index; i < edgeCount; i += stride) {
    // *** Get the index of the first point of the edge
    int pointIndexStart = tex1Dfetch<int>(edgeIndicesTex, static_cast<int>(i)),
        pointIndexEnd =
          i == edgeCount - 1
            ? static_cast<int>(pointCount)
            : tex1Dfetch<int>(edgeIndicesTex, static_cast<int>(i + 1));

    // *** Keep track of a counter for the current point index
    int pointIndex = pointIndexStart;

    // *** Get the length of the current edge
    auto edgeLength = tex1Dfetch<float>(edgeLengthsTex, static_cast<int>(i));

    // *** Keep looping until the last point is reached
    while (true) {
      auto point = tex1Dfetch<float2>(pointsTex, pointIndex++);

      // *** Check if the fetched point is the end of the line
      if (pointIndex > pointIndexEnd) {
        break;
      }

      // *** Calculate the index for the density image
      int siteIndex = static_cast<int>(point.y) * static_cast<int>(pitch) +
                      static_cast<int>(point.x);

      // *** Add the edge length to the density input
      densityOutput[siteIndex] += edgeLength;
    }
  }
}

__global__ void
generateDensityMapCount(uint* countsOutput,
                        hipTextureObject_t pointsTex,
                        hipTextureObject_t edgeIndicesTex,
                        size_t pointCount,
                        size_t edgeCount,
                        size_t pitch)
{
  // *** Do nothing if no countsOutput is specified
  if (!countsOutput) {
    return;
  }

  // *** Get the index and stride from gpu
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = blockDim.x * gridDim.x;

  // *** Loop over the edges
  for (size_t i = index; i < edgeCount; i += stride) {
    // *** Get the index of the first point of the edge
    int pointIndexStart = tex1Dfetch<int>(edgeIndicesTex, static_cast<int>(i)),
        pointIndexEnd =
          i == edgeCount - 1
            ? static_cast<int>(pointCount)
            : tex1Dfetch<int>(edgeIndicesTex, static_cast<int>(i + 1));

    // *** Keep track of a counter for the current point index
    int pointIndex = pointIndexStart;

    // *** Keep looping until the last point is reached
    while (true) {
      auto point = tex1Dfetch<float2>(pointsTex, pointIndex++);

      // *** Check if the fetched point is the end of the line
      if (pointIndex > pointIndexEnd) {
        break;
      }

      // *** Calculate the index for the density image
      int siteIndex = static_cast<int>(point.y) * static_cast<int>(pitch) +
                      static_cast<int>(point.x);

      // *** Add the edge length to the density input
      atomicAdd(&countsOutput[siteIndex], 1);
    }
  }
}

__global__ void
convertDensityMapToFloat(float* densityOutput,
                         hipTextureObject_t densityCountsTex,
                         int width,
                         int height)
{
  // *** Do nothing if no countsOutput is specified
  if (!densityOutput) {
    return;
  }

  // *** Get the index and stride from gpu
  size_t index_x = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride_x = blockDim.x * gridDim.x;
  size_t index_y = blockIdx.y * blockDim.y + threadIdx.y;
  size_t stride_y = blockDim.y * gridDim.y;

  for (size_t x = index_x; x < width; x += stride_x) {
    for (size_t y = index_y; y < height; y += stride_y) {
      densityOutput[y * width + x] =
        static_cast<float>(tex1Dfetch<uint>(densityCountsTex, y * width + x));
    }
  }
}

__global__ void
convolveDensityMap(float* output,
                   hipTextureObject_t densityTex,
                   hipTextureObject_t convolutionKernelTex,
                   int width,
                   int height,
                   size_t pitch,
                   int kernelRadius,
                   convolution_direction_t direction)
{
  size_t index_x = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride_x = blockDim.x * gridDim.x;
  size_t index_y = blockIdx.y * blockDim.y + threadIdx.y;
  size_t stride_y = blockDim.y * gridDim.y;

  for (size_t x = index_x; x < width; x += stride_x) {
    for (size_t y = index_y; y < height; y += stride_y) {
      const float pixel_x = static_cast<float>(x) + 0.5f;
      const float pixel_y = static_cast<float>(y) + 0.5f;

      float sum = 0;

      for (int k = -kernelRadius; k <= kernelRadius; ++k) {
        sum +=
          tex2D<float>(
            densityTex,
            pixel_x + static_cast<float>((direction == convolveRows) * k),
            pixel_y + static_cast<float>((direction == convolveColumns) * k)) *
          tex1Dfetch<float>(convolutionKernelTex, kernelRadius - k);
      }

      output[y * pitch + x] = sum;
    }
  }
}
} // namespace kernels

resource_2d<float>
gpu::generate_density_map(const linear_resource<glm::vec2>& pointsRes,
                          const linear_resource<int>& edgeIndicesRes,
                          const linear_resource<float>& edgeLengthsRes,
                          float kernelSize,
                          int resolution,
                          bool fastDensity)
{
  // *** Get the edge count from the size of the edge indices resource
  size_t pointCount = pointsRes.size(), edgeCount = edgeIndicesRes.size();

  // *** Create a resource for the density map
  resource_2d<float> densityMapRes(resolution, resolution);

  // *** Set to all zeros
  gpu_check hipMemset2D(densityMapRes.dev_ptr(),
                         densityMapRes.pitch() *
                           sizeof(decltype(densityMapRes)::value_type),
                         0x0,
                         densityMapRes.width(),
                         densityMapRes.height());

  {
    // *** If fast density is used, no atomic operations are used in the kernel
    // which will result in lower values than using the accurate approach
    if (fastDensity) {
      // *** Configure the kernel execution parameters
      size_t blockSize = 256;
      size_t numBlocks = (edgeCount + blockSize - 1) / blockSize;

      // *** Call the kernel
      kernels::generateDensityMapFast<<<blockSize, numBlocks>>>(
        densityMapRes.dev_ptr(),
        pointsRes.tex(),
        edgeIndicesRes.tex(),
        edgeLengthsRes.tex(),
        pointCount,
        edgeCount,
        densityMapRes.pitch());

      // *** Check kernel launch
      gpu_check hipPeekAtLastError();

      // *** Synchronise the kernel
      gpu_check hipDeviceSynchronize();
    } else {
      // *** Create a texture for keeping track of the density count
      resource_2d<uint> densityCountsRes(resolution, resolution);

      {
        // *** Configure the kernel execution parameters
        size_t blockSize = 256;
        size_t numBlocks = (edgeCount + blockSize - 1) / blockSize;

        // *** Set to all zeros
        gpu_check hipMemset2D(densityCountsRes.dev_ptr(),
                               densityCountsRes.pitch() *
                                 sizeof(decltype(densityMapRes)::value_type),
                               0x0,
                               densityCountsRes.width(),
                               densityCountsRes.height());

        kernels::generateDensityMapCount<<<blockSize, numBlocks>>>(
          densityCountsRes.dev_ptr(),
          pointsRes.tex(),
          edgeIndicesRes.tex(),
          pointsRes.size(),
          edgeIndicesRes.size(),
          densityCountsRes.pitch());

        gpu_check hipDeviceSynchronize();
      }

      {
        // *** Configure the kernel execution parameters
        dim3 blockSize(256, 256);
        dim3 numBlocks((resolution + blockSize.x - 1) / blockSize.x,
                       (resolution + blockSize.y - 1) / blockSize.y);

        kernels::convertDensityMapToFloat<<<blockSize, numBlocks>>>(
          densityMapRes.dev_ptr(),
          densityCountsRes.tex(),
          resolution,
          resolution);

        gpu_check hipDeviceSynchronize();
      }
    }
  }

  {
    // *** Generate the parabolic filter kernel
    const int kernelRadius = static_cast<int>(kernelSize / 2.0f) * 2;
    const int kernelLength = 2 * kernelRadius + 1;

    std::vector<float> h_parabolicFilterKernel;
    h_parabolicFilterKernel.reserve(kernelLength);

    for (size_t i = 0; i < kernelLength; i++) {
      auto x = static_cast<float>(i) / static_cast<float>(kernelLength - 1);
      x = std::abs(x - 0.5f) / 0.5f;
      h_parabolicFilterKernel.emplace_back(1.0f - x * x);
    }

    // *** Copy the kernel to a texture
    linear_resource parabolicFilterKernelRes(h_parabolicFilterKernel);

    // *** Configure the kernel execution parameters
    dim3 blockSize(256, 256);
    dim3 numBlocks((resolution + blockSize.x - 1) / blockSize.x,
                   (resolution + blockSize.y - 1) / blockSize.y);

    {
      // *** Create a resource for the vertically convoluted density map
      resource_2d<float> convDensityMapRes(resolution, resolution);

      kernels::convolveDensityMap<<<blockSize, numBlocks>>>(
        convDensityMapRes.dev_ptr(),
        densityMapRes.tex(),
        parabolicFilterKernelRes.tex(),
        resolution,
        resolution,
        convDensityMapRes.pitch(),
        kernelRadius,
        convolveRows);

      // *** Check kernel launch
      gpu_check hipPeekAtLastError();

      // *** Synchronise the kernel
      gpu_check hipDeviceSynchronize();

      // *** Move the convoluted density map over to the density map
      densityMapRes = std::move(convDensityMapRes);
    }

    {
      // *** Create a resource for the vertically convoluted density map
      resource_2d<float> convDensityMapRes(resolution, resolution);

      kernels::convolveDensityMap<<<blockSize, numBlocks>>>(
        convDensityMapRes.dev_ptr(),
        densityMapRes.tex(),
        parabolicFilterKernelRes.tex(),
        resolution,
        resolution,
        convDensityMapRes.pitch(),
        kernelRadius,
        convolveColumns);

      // *** Check kernel launch
      gpu_check hipPeekAtLastError();

      // *** Synchronise the kernel
      gpu_check hipDeviceSynchronize();

      // *** Move the convoluted density map over to the density map
      densityMapRes = std::move(convDensityMapRes);
    }
  }

  return densityMapRes;
}
} // namespace cubu::internal