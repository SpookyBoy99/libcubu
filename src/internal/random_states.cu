#include "hip/hip_runtime.h"
#include "cubu/internal/gpu_check.hpp"
#include "cubu/internal/random_states.hpp"

namespace cubu::internal {
namespace kernels {
__global__ void
initRandomStates(hiprandState* states, size_t size)
{
  // *** Get the index and stride from gpu
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = blockDim.x * gridDim.x;

  // *** Init the array of random states
  for (size_t i = index; i < size; i += stride) {
    hiprand_init(1234, i, 0, &states[i]);
  }
}
} // namespace kernels

random_states::random_states(size_t size)
  : d_randomStates_(nullptr)
  , size_(size)
{
  // *** Initialize the random states
  gpu_check hipMalloc((void**)&d_randomStates_,
                       size_ * sizeof(d_randomStates_[0]));

  // *** Call the kernel
  kernels::initRandomStates<<<1, size_>>>(d_randomStates_, size_);

  // *** Check kernel launch
  gpu_check hipPeekAtLastError();

  // *** Synchronise the kernel
  gpu_check hipDeviceSynchronize();
}

random_states::~random_states()
{
  gpu_check hipFree(d_randomStates_);
}

hiprandState*
random_states::data() const
{
  return d_randomStates_;
}

size_t
random_states::size() const
{
  return size_;
}
} // namespace cubu::internal