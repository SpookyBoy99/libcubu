#include "cubu/internal/gpu_check.hpp"
#include "cubu/internal/kernels.hpp"
#include "cubu/internal/random_states.hpp"

namespace cubu::internal {
random_states::random_states(size_t size)
  : d_randomStates_(nullptr)
  , size_(size)
{
  // *** Initialize the random states
  gpu_check hipMalloc((void**)&d_randomStates_,
                       size_ * sizeof(d_randomStates_[0]));

  // *** Call the kernel
  kernels::initRandomStates<<<1, size_>>>(d_randomStates_, size_);

  // *** Check kernel launch
  gpu_check hipPeekAtLastError();

  // *** Synchronise the kernel
  gpu_check hipDeviceSynchronize();
}

random_states::~random_states()
{
  gpu_check hipFree(d_randomStates_);
}

hiprandState*
random_states::data() const
{
  return d_randomStates_;
}

size_t
random_states::size() const
{
  return size_;
}
} // namespace cubu::internal